
#include <hip/hip_runtime.h>
extern "C"
__global__ 
void 
k0(
    float *in,
    float *out,
    size_t ldN,
    float f1,
    float f2,
    float f3
) {
    size_t COL_0 = 0;
    size_t COL_1 = ldN;
    size_t COL_2 = 2*ldN;

    if (threadIdx.x < ldN) {
        out[COL_0 + threadIdx.x] = f3;
        out[COL_1 + threadIdx.x] = f2;

        out[COL_2 + threadIdx.x] = in[COL_0 + threadIdx.x] + 99.0f;
    }
}
